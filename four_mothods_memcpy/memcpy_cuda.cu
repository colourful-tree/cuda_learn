#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

int N = 64; 

__global__ void VecAddOne(int* A) {
    int i = threadIdx.x;
    A[i] = A[i] + 1;
}


bool test_cudaHostAlloc() {

  bool res = true;
  int *dptr1, *hptr1;

  auto start = std::chrono::system_clock::now();

  // First way: using cudaHostAlloc()
  // Alloc host page-locked memory
  (hipHostAlloc(&hptr1, sizeof(int) * N, hipHostMallocMapped));

  // Get corresponding device pointer
  (hipHostGetDevicePointer(&dptr1, hptr1, 0));

  // Initialize this memory
  for(int i = 0; i < N; i++)
    hptr1[i] = i;

  // Test kernel
  VecAddOne<<<1, N>>>(dptr1);
  hipDeviceSynchronize();

  // Check result
  for(int i = 0; i < N; i++){
    //printf("s[%d]: %d\n", i, hptr1[i]);
    res = (hptr1[i] == i + 1) ? res : false;
  }
  // Free memory
  (hipHostFree(hptr1));
  auto end = std::chrono::system_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "hipHostAlloc time:" << duration.count() << std::endl;

  return res;
}

bool test_cudaHostRegister() {

  bool res = true;
  int *dptr1, *hptr1;

  // Second way: using cudaHostRegister()
  // Alloc host memory
  hptr1 = new int[N];
  assert(hptr1);

  // Initialize this memory
  for(int i = 0; i < N; i++)
    hptr1[i] = i;

  auto start = std::chrono::system_clock::now();
  // Page-lock host memory
  hipHostRegister(hptr1, N, hipHostRegisterMapped);

  // Get corresponding device pointer
  (hipHostGetDevicePointer(&dptr1, hptr1, 0));

  // Test kernel
  if(hipDeviceAttributeCanUseHostPointerForRegisteredMem != 0){
    printf("Can directly use host pointer to substitute device pointer on this machine.\n");
    VecAddOne<<<1, N>>>(hptr1);
  }
  else{
    printf("This machine does not support substituting host pointer for device pointer.\n");
    VecAddOne<<<1, N>>>(dptr1);
  }
  hipDeviceSynchronize();
  // Free memory
  hipHostUnregister(hptr1);
  auto end = std::chrono::system_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "hipHostRegister time:" << duration.count() << std::endl;

  // Check result
  for(int i = 0; i < N; i++){
    //printf("s[%d]: %d\n", i, hptr1[i]);
    res = (hptr1[i] == i + 1) ? res : false;
  }
  delete hptr1;
  return res;
}

bool test_cudaMemcpy() {

  bool res = true;
  int *dptr1, *hptr1;

  // Alloc host memory
  hptr1 = new int[N];
  assert(hptr1);

  // Initialize this memory
  for(int i = 0; i < N; i++)
    hptr1[i] = i;

  auto start = std::chrono::system_clock::now();
  hipMalloc(&dptr1, sizeof(int) * N);
  hipMemcpy(dptr1, hptr1, sizeof(int) * N, hipMemcpyHostToDevice);

  VecAddOne<<<1, N>>>(dptr1);
  hipDeviceSynchronize();
  hipMemcpy(hptr1, dptr1, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipFree(dptr1);

  auto end = std::chrono::system_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "hipMemcpy time:" << duration.count() << std::endl;

  // Check result
  for(int i = 0; i < N; i++){
    // printf("s[%d]: %d\n", i, hptr1[i]);
    res = (hptr1[i] == i + 1) ? res : false;
  }

  // Free memory
  delete hptr1;
  return res;
}

bool test_cudaMemcpyAsync() {

  bool res = true;
  int *dptr1, *hptr1;

  // Alloc host memory
  hptr1 = new int[N];
  assert(hptr1);

  // Initialize this memory
  for(int i = 0; i < N; i++)
    hptr1[i] = i;
  
  auto start = std::chrono::system_clock::now();

  hipMalloc(&dptr1, sizeof(int) * N);
  hipMemcpyAsync(dptr1, hptr1, sizeof(int) * N, hipMemcpyHostToDevice);

  VecAddOne<<<1, N>>>(dptr1);
  hipDeviceSynchronize();
  hipMemcpyAsync(hptr1, dptr1, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipFree(dptr1);

  auto end = std::chrono::system_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "hipMemcpyAsync time:" << duration.count() << std::endl;

  // Check result
  for(int i = 0; i < N; i++){
    //printf("s[%d]: %d\n", i, hptr1[i]);
    res = (hptr1[i] == i + 1) ? res : false;
  }

  // Free memory
  delete hptr1;
  return res;
}

int main() {
  // Enable memory mapping
  // cudaSetDeviceFlags(cudaDeviceMapHost);
  // cout << test_cudaHostAlloc() << endl; //First will take long time, it's little confused.

  cout << test_cudaHostAlloc() << endl;
  cout << test_cudaHostRegister() << endl;
  cout << test_cudaMemcpy() << endl;
  cout << test_cudaMemcpyAsync() << endl;

  return 0;
}
